#include "utils/StatevectorCUDA.h"

#include <hip/hip_runtime.h>

#define CALL_CU(FUNC, MSG) \
  cuResult = FUNC; \
  if (cuResult != hipSuccess) { \
    std::cerr << RED("[CUDA Err] ") << MSG << ". Error code " \
              << cuResult << "\n"; \
    return; \
  }

#define CALL_CUDA(FUNC, MSG) \
  cudaResult = FUNC; \
  if (cudaResult != hipSuccess) { \
    std::cerr << RED("[CUDA Err] ") << MSG << ". " \
              << hipGetErrorString(cudaResult) << "\n"; \
    return; \
  }

using namespace utils;

template<typename ScalarType>
void StatevectorCUDA<ScalarType>::mallocDeviceData() {
  assert(dData == nullptr && "Already allocated");
  CUDA_CALL(hipMalloc(&dData, sizeInBytes()),
    "Failed to allocate memory for statevector on the device");
  syncState = DeviceIsNewer;
}

template<typename ScalarType>
void StatevectorCUDA<ScalarType>::freeDeviceData() {
  CUDA_CALL(hipFree(dData),
    "Failed to free memory for statevector on the device");
  syncState = HostIsNewer;
}

template<typename ScalarType>
void StatevectorCUDA<ScalarType>::initialize() {
  if (dData == nullptr)
    mallocDeviceData();
  CALL_CUDA(hipMemset(dData, 0, sizeInBytes()),
    "Failed to zero statevector on the device");
  ScalarType one = 1.0;
  CALL_CUDA(hipMemcpy(dData, &one, sizeof(ScalarType), hipMemcpyHostToDevice),
    "Failed to set the first element of the statevector to 1");
  syncState = DeviceIsNewer;
}

template class StatevectorCUDA<float>;
template class StatevectorCUDA<double>;

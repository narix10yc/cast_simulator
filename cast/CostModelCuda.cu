#include "hip/hip_runtime.h"
#include "cast/CostModel.h"

using namespace cast;

// Definition of the dummy kernel
__global__ void emptyKernel() {}

// Definition of the member function
void CUDACostModel::measureLaunchOverheadOnce() {
    constexpr int N = 10000;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < N; ++i) {
        emptyKernel<<<1,1>>>();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float msElapsed = 0.0f;
    hipEventElapsedTime(&msElapsed, start, stop);

    // Average overhead per kernel launch (seconds)
    measuredLaunchOverhead = (msElapsed / 1e3) / double(N);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cerr << "Measured launch overhead: "
              << measuredLaunchOverhead << " s\n";
}
